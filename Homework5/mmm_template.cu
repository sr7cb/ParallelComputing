// Performance based GPU implementation of Matrix Multiply using NVIDIA CUDA Programming Language
//Sanil Rao 5/8/17 CS4444


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>
#include<stdlib.h>
#include<iostream>

//number of threads per block sent to the kernel
#define threads_per_block 2
using namespace std;

//----------------------------------- Structures and Globals---------------------------------------------
typedef struct {
	int dimension1;
	int dimension2;	
} ArrayMetadata2D;

// metadata variables describing dimensionalities of all data structures involved in the computation
ArrayMetadata2D A_MD, B_MD, C_MD;
// pointers for input and output arrays in the host memory  
float *A, *B, *C, *C_CPU;
// pointers for input and output arrays in the device memory (NVIDIA DRAM)
float *A_GPU, *B_GPU, *C_GPU;

//----------------------------------- host function definitions -----------------------------------------

void allocateAndInitializeAB();
void computeCpuMMM();
void copyMatricesToGPU();
void copyResultFromGPU();
void compareHostAndGpuOutput();
void die(const char *error); 
void check_error(hipError_t e);

//----------------------------------- CUDA function definitions -----------------------------------------
//barebones GPU implementation kernel
void GPU_matrix_multiply();
//optimized GPU implementation kernel
void GPU_matrix_multiply_opt();

//-------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {
	
	A_MD.dimension1 = (argc > 1) ? atoi(argv[1]) : 100;
	A_MD.dimension2 = (argc > 2) ? atoi(argv[2]) : A_MD.dimension1;
	B_MD.dimension1 = (argc > 3) ? atoi(argv[3]) : A_MD.dimension2;
	B_MD.dimension2 = (argc > 4) ? atoi(argv[4]) : B_MD.dimension1;
	C_MD.dimension1 = A_MD.dimension1;
	C_MD.dimension2 = B_MD.dimension2;

	printf("Matrix A is %d-by-%d\n", A_MD.dimension1, A_MD.dimension2);
	printf("Matrix B is %d-by-%d\n", B_MD.dimension1, B_MD.dimension2);
	printf("Matrix C is %d-by-%d\n", C_MD.dimension1, C_MD.dimension2);

	allocateAndInitializeAB();

	//Timing the CPU implementation that was unmodified

	// matrix matrix multiplication in the CPU
	//	clock_t start = clock();	
	//	computeCpuMMM();
	//	clock_t end = clock();
   
	//	double elapsed = (end - start) / (double) CLOCKS_PER_SEC;
	//	printf("Computation time in the CPU: %f seconds\n", elapsed);
	
	//Timing the barebones GPU implementation
	
	//clock_t startGPU = clock();
	//GPU_matrix_multiply();	
	//clock_t endGPU = clock();

	//Timing the optimized GPU implementation

	clock_t startGPU = clock();
	GPU_matrix_multiply_opt();	
	clock_t endGPU = clock();
	double elapsedGPU = (endGPU-startGPU)/ (double) CLOCKS_PER_SEC;
	printf("Computation time in the GPU: %f seconds\n", elapsedGPU);	
	//double elapsedGPU2 = (endGPU2-startGPU1)/ (double) CLOCKS_PER_SEC;
	//printf("Computation time in the GPU: %f seconds\n", elapsedGPU2);

	//compareHostAndGpuOutput();	
	return 0;
}

//Optimzed GPU kernel using tiling, shared memory, and transposed matricies
//2 dimensional grid and block size to ease with computation overhead I think better in 2D for a 2D problem
__global__ void mm_kernel_opt(float *A_GPU,float *B_GPU, float * C_GPU, ArrayMetadata2D A_MD) {
	//determing which block that is being computed
	 int block_id_row = blockIdx.y;
	 int block_id_col = blockIdx.x;

	 //resultant value to placed into the output matrix
	 float val;
	 val = 0;

	//determing specific row and column value for each thread
	 int row = threadIdx.y;
	 int col = threadIdx.x;
		
	//declaring shared memory to be used during computation
	__shared__ float  A[threads_per_block][threads_per_block];
	__shared__ float  B[threads_per_block][threads_per_block];

	//looping over the size of each tile and computnig the value
	 for(int j = 0; j < (threads_per_block + A_MD.dimension1 -1)/threads_per_block; j++) {

			//thread out of bounds check
			if((j * threads_per_block + col) < A_MD.dimension1 &&(block_id_row*threads_per_block+row) < A_MD.dimension1) 
				//intital values to be placed into shared memory
				A[row][col] = A_GPU[(block_id_row*threads_per_block + row) * A_MD.dimension1 + j*threads_per_block + col];
			else
				//excess values used to not impact computation
				A[row][col] = 0.0;
			if((j*threads_per_block + row) < A_MD.dimension1 && (block_id_col*threads_per_block+col) < A_MD.dimension1) 
				//intital values to be placed into shared memory this case transposed 
				B[row][col] = B_GPU[(block_id_col*threads_per_block+col) * A_MD.dimension1 +(j*threads_per_block + row)];
			else
				//excess values used to not impact computation
				B[row][col] = 0.0;
		
		//barrier to make sure copying was completed before computing
		__syncthreads();

		//computation phase
		for(int c = 0; c < threads_per_block; c++) {
			val += A[row][c] * B[c][col];
		}
		//barrier to make sure computation phase was done correctly
		__syncthreads();
		
		//final bounds check 
		if((block_id_row*threads_per_block+row) < A_MD.dimension1 && (block_id_col*threads_per_block + col) < A_MD.dimension1)
			//placing the value into the output matrix
			C_GPU[((block_id_row * blockDim.y + row) * A_MD.dimension1) + (block_id_col * blockDim.x) + col] = val;

	}
}

//barebones GPU kernel 1 dimension grid and block size
__global__ void mm_kernel(float *A_GPU, float*B_GPU, float *C_GPU, ArrayMetadata2D A_MD) {

	//definitions to aid in GPU programming SR 4/30/17
	//block id gives value of each indivdual block which together make the whole grid
	//block dim gives the value of the block size
	// thread id which gives each thread within the block. 

	 //determing each threads block and thread number
	 int block_id = blockIdx.x;
     int global_thread_id = blockDim.x * block_id + threadIdx.x;

	 int k, i;
	 float val;
	
	//computing over the matricies using the global thread number as the column as it was inputed in 1 dimension
	 for(i = 0; i < A_MD.dimension1; i++) {
		val = 0;
		for(k = 0; k < A_MD.dimension2; k++) {
			 val += A_GPU[i*A_MD.dimension2 + k] * B_GPU[k * A_MD.dimension2 + global_thread_id];
			 C_GPU[i*A_MD.dimension2 + global_thread_id] = val;
		 }
	 }
}

//host code to launch the optimized kernel
void GPU_matrix_multiply_opt() {
	copyMatricesToGPU();
	dim3 block_size(threads_per_block, threads_per_block);	
	dim3 grid_size(ceil(((float)A_MD.dimension1)/threads_per_block),ceil(((float)A_MD.dimension1)/threads_per_block));
	mm_kernel_opt<<<grid_size, block_size>>> (A_GPU, B_GPU, C_GPU, A_MD);
	copyResultFromGPU();
}

//  host code to launch the base kernel 
void GPU_matrix_multiply() {
	copyMatricesToGPU();
	dim3 grid_size((A_MD.dimension2 + threads_per_block*threads_per_block -1)/(threads_per_block*threads_per_block));
	dim3 block_size(threads_per_block * threads_per_block);
	mm_kernel <<<grid_size, block_size>>> (A_GPU,B_GPU,C_GPU,A_MD);
	copyResultFromGPU();
}

// allocate and initialize A and B using a random number generator
void allocateAndInitializeAB() {
	
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	A = (float*) malloc(sizeofA);
	
	srand(time(NULL));
  	for (int i = 0; i < A_MD.dimension1; i++) {
		for (int j = 0; j < A_MD.dimension2; j++) {
			int index = i * A_MD.dimension2 + j;
			A[index] = (rand() % 1000) * 0.001; 
		}
	}
			
	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	B = (float*) malloc(sizeofB);
  	for (int i = 0; i < B_MD.dimension1; i++) {
		for (int j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			B[index] = (rand() % 1000) * 0.001; 
		}
	}
}

// allocate memory in the GPU for all matrices, and copy A and B content from the host CPU memory to the GPU memory
//if barebones kernel is to be launched dont transpose the matrix comment out the transpose
void copyMatricesToGPU() {
	
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &A_GPU, sizeofA));
	check_error(hipMemcpy(A_GPU, A, sizeofA, hipMemcpyHostToDevice));
	
	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &B_GPU, sizeofB));
	int i,j;
	/*for(i = 0; i < B_MD.dimension1; i++) {
		for(j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			printf("%f  ", B[index]);
		}
		printf("\n");
	}*/
	for(i = 0; i < B_MD.dimension1; i++) {
			int fi = i * B_MD.dimension2;
		for(j = 0; j < B_MD.dimension2; j++) {
			if(j > i) {
			int index = fi +j;
			int newindex = j * B_MD.dimension2 + i;
			float tmp = B[index];
			B[index] = B[newindex];
			B[newindex] = tmp;
			}
		}
	} 
	/*for(i = 0; i < B_MD.dimension1; i++) {
		for(j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			printf("%f  ", B[index]);
		}
		printf("\n");
	}*/
	check_error(hipMemcpy(B_GPU, B, sizeofB, hipMemcpyHostToDevice));
	
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &C_GPU, sizeofC));
}

// copy results from C_GPU which is in GPU card memory to C_CPU which is in the host CPU for result comparison
void copyResultFromGPU() {
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C_CPU = (float*) malloc(sizeofC);
	check_error(hipMemcpy(C_CPU, C_GPU, sizeofC, hipMemcpyDeviceToHost));
}

// do a straightforward matrix-matrix multiplication in the CPU
// notice that this implementation can be massively improved in the CPU by doing proper cache blocking but we are
// not providing you the efficient CPU implementation as that reveals too much about the ideal GPU implementation
void computeCpuMMM() {
	
	// allocate the result matrix for the CPU computation
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C = (float*) malloc(sizeofC);
	
	// compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
	for (int i = 0; i < A_MD.dimension1; i++) {
		int a_i = i * A_MD.dimension2;
		int c_i = i * C_MD.dimension2;
		for (int j = 0; j < B_MD.dimension2; j++) {
			int c_index = c_i + j;
			C[c_index] = 0;
			for (int k = 0; k < B_MD.dimension1; k++) {
				int a_index = a_i + k;
				int b_index = k * B_MD.dimension2 + j;
				C[c_index] += A[a_index] * B[b_index];
			}
		}
	}
}

// function to determine if the GPU computation is done correctly by comparing the output from the GPU with that
// from the CPU
void compareHostAndGpuOutput() {
	int totalElements = C_MD.dimension1 * C_MD.dimension2;
	int missmatchCount = 0;
	for (int i = 0; i < totalElements; i++) {
		if (fabs(C[i] - C_CPU[i]) > 0.01) {
			missmatchCount++;
			printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
		}
	}
	if (missmatchCount > 0) {
		printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
	} else {
		printf("Computation is correct: CPU and GPU outputs match\n");
	}
}

// Prints the specified error message and then exits
void die(const char *error) {
        printf("%s", error);
        exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
        if (e != hipSuccess) {
                printf("\nCUDA error: %s\n", hipGetErrorString(e));
                exit(1);
        }
}

